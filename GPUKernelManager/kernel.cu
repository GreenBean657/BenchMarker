#include "kernel.h"
#include <hip/hip_runtime.h>

namespace GPUKernel {
    void* tryAlloc(std::size_t arrBytes) {
        void* ptr = nullptr;
        hipMalloc(&ptr, arrBytes);
        return ptr;
    }
}